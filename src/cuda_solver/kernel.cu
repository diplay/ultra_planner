#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
//#include <core/transition_system/UState.h>
//#include "CUState.h"
#include <core/transition_system/UTransitionSystem.h>
#include <stdio.h>
#include <vector>
#include "kernel.cuh"
#include "helpers.cuh"
#include "cuPrintf.cuh"
#include "CUBitset.cuh"
#include "CUState.cuh"
#include "CUTransition.cuh"

__device__ bool transition_is_active(const CUState & state, const CUTransition & transition)
{
	bool r = state.flags.equalMasked(transition.condition.flags.state, transition.condition.flags.mask);
	if(!r)
		return false;

	r = state.flags.equalMasked(transition.effect.flags.state, transition.effect.flags.mask);
	if(r)
		return false;

	return true;
}

//__constant__ char cTransitionsBuffer[40000];

//Create memory for transitions and copy all transitions
__host__ void save_transitions(const UTransitionSystem * transition_system, UCUDATaskContext * task_context)
{
	hipError_t r = hipSuccess;

	//Calculate maximum transition size
	size_t max_transition_size = 0;
	for(auto tr : transition_system->transitions())
		max_transition_size = max(max_transition_size, tr.plainDataSize());
	task_context->transitionSize = align_memory_size<4>(max_transition_size);

	//Allocate memory for all transitions
	const size_t data_size = task_context->transitionSize * transition_system->transitions().size();
	r = hipMalloc(&(task_context->transitionsData), data_size);
	printf("Total transitions size: %d bytes.\n", data_size);

	//Copy them to GPU
	char * transitions_data = (char*)malloc(data_size);
	for(int i = 0; i < transition_system->transitions().size(); ++i)
	{
		auto tr = transition_system->transition(i);
		size_t tr_size = tr.serialize(transitions_data + i*task_context->transitionSize);
	}

	r = hipMemcpy(task_context->transitionsData, transitions_data, data_size, hipMemcpyHostToDevice);
	//hipMemcpyToSymbol(HIP_SYMBOL(cTransitionsBuffer), transitions_data, data_size);

	free(transitions_data);
}

/*
Expands states in parallel where:
- each state to expand is a block
- each transition is a thread
*/
__global__ void expand_state(void * states_buffer, size_t state_size, void * transitions_buffer, const size_t transition_size, unsigned int * result_state_count, int * source_states, int * applied_transitions, char * result_states_data)
{
	//Get current state
	size_t state_index = blockIdx.x;
	void * state_addr = (char*)states_buffer + state_index * state_size;
	CUState state(state_addr);

	//Get current transition
	size_t transition_index = threadIdx.x;
	void * transition_addr = (char*)transitions_buffer + transition_index * transition_size;
	//void *transition_addr = cTransitionsBuffer + + transition_index * transition_size;
	CUTransition transition(transition_addr);
	//transition.print();

	//Check if transition is active
	if(transition_is_active(state, transition))
	{
		//printf("Active!\n");
		//Increase result state count
		unsigned int new_state_index = atomicInc(result_state_count, UINT32_MAX);

		//Create new state
		char * new_state_data_addr = result_states_data + new_state_index * state_size;

		size_t ss = state.serialize(new_state_data_addr);
		CUState new_state;
		size_t ds = new_state.deserialize(new_state_data_addr);
		
		//Apply transition and automatically write new state to buffer
		new_state.apply(transition);
	
		//Write used transition to buffer
		applied_transitions[new_state_index] = transition_index;

		//Write source state index to buffer
		source_states[new_state_index] = state_index;
	}
}

__host__ ExpansionResult expand_states(UCUDATaskContext * task_context, size_t state_count, char *& source_states_buffer)
{
	//=========Create memory bufferof on GPU=============
	//Result counter
	unsigned int * cu_res_state_count;
	hipMalloc(&cu_res_state_count, sizeof(unsigned int));
	hipMemset(cu_res_state_count, 0, sizeof(unsigned int));

	const size_t max_result_count = task_context->transitionCount * state_count;
	//Result states data
	char * cu_states_buffer;
	hipMalloc(&cu_states_buffer, max_result_count * task_context->stateSize);

	//Result source state indices
	int * cu_source_states;
	hipMalloc(&cu_source_states, max_result_count * sizeof(int));

	//Applied transition index
	int * cu_applied_transitions;
	hipMalloc(&cu_applied_transitions, max_result_count * sizeof(int));
	
	//Run kernel
	//
	expand_state<<<state_count, task_context->transitionCount>>>(source_states_buffer, task_context->stateSize, task_context->transitionsData, task_context->transitionSize, cu_res_state_count, cu_source_states, cu_applied_transitions, cu_states_buffer);

	int result_state_count;
	hipMemcpy(&result_state_count, cu_res_state_count, sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);

	//printf("After expansion got %d nodes.\n", result_state_count);

	//=================Copy results back to RAM==================
	int * source_states = (int*)malloc(result_state_count * sizeof(int));
	int * applied_transitions = (int*)malloc(result_state_count * sizeof(int));
	char * states_buffer = (char*)malloc(max_result_count * task_context->stateSize);

	hipMemcpy(source_states, cu_source_states, result_state_count * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);
	hipMemcpy(applied_transitions, cu_applied_transitions, result_state_count * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost);
	hipMemcpy(states_buffer, cu_states_buffer, result_state_count * task_context->stateSize, hipMemcpyKind::hipMemcpyDeviceToHost);

	
	ExpansionResult res;
	res.elements = new ExpansionElement[result_state_count];
	res.count = result_state_count;
	//res.reserve(result_state_count);

	//Deserialize result
	for(int i = 0; i < result_state_count; ++i)
	{
		ExpansionElement * el = res.elements + i;
		el->sourceStateIndex = source_states[i];
		el->transitionIndex = applied_transitions[i];
		el->state.deserialize(states_buffer + task_context->stateSize * i);
	}

	//Clear RAM
	free(source_states);
	free(applied_transitions);
	free(states_buffer);

	//Clear CUDA memory
	hipFree(cu_res_state_count);
	hipFree(cu_states_buffer);
	hipFree(cu_source_states);
	hipFree(cu_applied_transitions);

	return std::move(res);
}
