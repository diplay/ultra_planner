
#include "helpers.cuh"
#include <stdio.h>
#include <string>
#include <stdint.h>
/*
using namespace CudaSolver;

__device__ size_t deserialize_int(char * data, int & value)
{
	int32_t val;
	memcpy(&val, data, sizeof(int32_t));
	value = val;
	return sizeof(int32_t);
}

__device__ size_t cu_serialize_int(char * dest, const int32_t val)
{
	memcpy(dest, &val, sizeof(val));
	return sizeof(int32_t);
}
*/

__device__ size_t cu_deserialize_int(char * data, int & value)
{
	int32_t val;
	memcpy(&val, data, sizeof(int32_t));
	value = val;
	return sizeof(int32_t);
}

__device__ size_t cu_serialize_int(char * dest, const int32_t val)
{
	memcpy(dest, &val, sizeof(val));
	return sizeof(int32_t);
}
